#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <cinttypes>
#include <NvInfer.h>
#include "plugin_audio_sb.h"

#define BLOCK_SIZE 128
__global__ void kernel_audio_sb(float* output, float* input, float* audio_0, float* audio_1, int S, int E, int tcount)
{
	int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (pos >= tcount) return;

	int e_idx = pos % E;
	int idx = pos / E;
	int s_idx = idx % S;
	int b_idx = idx / S;

	if (s_idx < S / 2) {
		int source_idx = b_idx*S/2*E + s_idx*E +e_idx;
		output[pos] = audio_0[source_idx];
	}
	else {
		int b_val_idx = b_idx * S/2 * E + (s_idx-S/2) * E + e_idx;
		float b_val = audio_1[b_val_idx] - input[b_val_idx];
		float s_val = expf(-input[pos]);
		output[pos] = b_val * s_val;
	}
}

void cuda_audio_sb(float* output, float* input, float* audio_0, float* audio_1, int N, int S, int E, hipStream_t stream)
{
	int tcount = N * S * E;
	int block = BLOCK_SIZE; // 128
	int grid = (tcount + block - 1) / block;
	//int shm_bytes = (E + block / 2 + block / 2) * sizeof(float); // sum, sum2
	kernel_audio_sb << < grid, block, 0, stream >> > (output, input, audio_0, audio_1, S, E, tcount);
}